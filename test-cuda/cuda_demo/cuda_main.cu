#include <iostream>
#include "hip/hip_runtime.h"

using namespace std;

__global__ void MatMul_gpu(int n, int *a, int *b, int *c) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;//x����ֵ
	int j = blockIdx.y * blockDim.y + threadIdx.y;//y����ֵ

	if (i < n && j < n) {
		int temp = 0;
		for (int k = 0; k < n; ++k) {
			temp += *(a + i * n + k) * *(b + k * n + j);
		}
		*(c + i * n + j) = temp;
	}
}

void MatMul_cpu(int n, int* a, int* b, int* c) {
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			*(c + i * n + j) = 0;
			for (int k = 0; k < n; ++k) {
				*(c + i * n + j) += *(a + i * n + k) * *(b + k * n + j);
			}
		}
	}
}

int main() {
	int const N = 5000;
	int* a, * b, * c;
	int* p_a, * p_b, * p_c;
	//����CPU���Դ�
	a = (int*)malloc(sizeof(int) * N * N);
	b = (int*)malloc(sizeof(int) * N * N);
	c = (int*)malloc(sizeof(int) * N * N);
	//����GPU���Դ�
	hipMalloc((void**)&p_a, sizeof(int) * N * N);
	hipMalloc((void**)&p_b, sizeof(int) * N * N);
	hipMalloc((void**)&p_c, sizeof(int) * N * N);
	//��ʼ������
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
			*(a + i * N + j) = (int)(rand() % 10 + 1);
			*(b + i * N + j) = (int)(rand() % 10 + 1);
			*(c + i * N + j) = 0;
		}
	}
	//��ӡ���˾���
	//cout << "A:" << endl;
	//for (int i = 0; i < N; ++i) {
	//	for (int j = 0; j < N; ++j) {
	//		cout << *(a + i * N + j) << "  ";
	//	}
	//	cout << endl;
	//}
	//cout << "B:" << endl;
	//for (int i = 0; i < N; ++i) {
	//	for (int j = 0; j < N; ++j) {
	//		cout << *(b + i * N + j) << "  ";
	//	}
	//	cout << endl;
	//}
	//�����Դ�
	hipMemcpy(p_a, a, sizeof(int) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(p_b, b, sizeof(int) * N * N, hipMemcpyHostToDevice);
	//����gpu�߳̿鼰�߳�������֯��ʽ
	dim3 dimGrid(ceil(N / 4.0), ceil(N / 4.0), 1);
	dim3 dimBlock(4, 4, 1);

	//����event�¼�������ִ������ӱ�ǵ㣬��ӡgpuִ��ʱ��
	//cudaEvent_t start1, stop1;
	//cudaEventCreate(&start1);
	//cudaEventCreate(&stop1);
	//cudaEventRecord(start1, 0);
	//MatMul_gpu <<<dimGrid, dimBlock>>> (N, p_a, p_b, p_c);//�˺�������
	//cudaEventRecord(stop1, 0);
	//cudaEventSynchronize(stop1);
	//float time1;
	//cudaEventElapsedTime(&time1, start1, stop1);
	//cout << "gpu time = " << time1 << endl;
	//����������
	//cudaMemcpy(c, p_c, sizeof(int) * N * N, cudaMemcpyDeviceToHost);

	//��ӡcpuִ��ʱ��
	hipEvent_t start2, stop2;
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
	hipEventRecord(start2, 0);
	MatMul_cpu(N, a, b, c);//��������
	hipEventRecord(stop2, 0);
	hipEventSynchronize(stop2);
	float time2;
	hipEventElapsedTime(&time2, start2, stop2);
	cout << "cpu time = " << time2 << endl;

	//��ӡ������
	//cout << "A*B:" << endl;
	//for (int i = 0; i < N; ++i) {
	//	for (int j = 0; j < N; ++j) {
	//		cout << *(c + i * N + j) << "  ";
	//	}
	//	cout << endl;
	//}
	//�����ڴ�
	free(a);
	free(b);
	free(c);
	hipFree(p_a);
	hipFree(p_b);
	hipFree(p_c);
	return 0;
}
#include <iostream>
#include "hip/hip_runtime.h"

using namespace std;

__global__ void MatMul_gpu(int n, int *a, int *b, int *c) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;//x方向值
	int j = blockIdx.y * blockDim.y + threadIdx.y;//y方向值

	if (i < n && j < n) {
		int temp = 0;
		for (int k = 0; k < n; ++k) {
			temp += *(a + i * n + k) * *(b + k * n + j);
		}
		*(c + i * n + j) = temp;
	}
}

void MatMul_cpu(int n, int* a, int* b, int* c) {
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			*(c + i * n + j) = 0;
			for (int k = 0; k < n; ++k) {
				*(c + i * n + j) += *(a + i * n + k) * *(b + k * n + j);
			}
		}
	}
}

int main() {
	int const N = 5000;
	int* a, * b, * c;
	int* p_a, * p_b, * p_c;
	//分配CPU端显存
	a = (int*)malloc(sizeof(int) * N * N);
	b = (int*)malloc(sizeof(int) * N * N);
	c = (int*)malloc(sizeof(int) * N * N);
	//分配GPU端显存
	hipMalloc((void**)&p_a, sizeof(int) * N * N);
	hipMalloc((void**)&p_b, sizeof(int) * N * N);
	hipMalloc((void**)&p_c, sizeof(int) * N * N);
	//初始化矩阵
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
			*(a + i * N + j) = (int)(rand() % 10 + 1);
			*(b + i * N + j) = (int)(rand() % 10 + 1);
			*(c + i * N + j) = 0;
		}
	}
	//打印待乘矩阵
	//cout << "A:" << endl;
	//for (int i = 0; i < N; ++i) {
	//	for (int j = 0; j < N; ++j) {
	//		cout << *(a + i * N + j) << "  ";
	//	}
	//	cout << endl;
	//}
	//cout << "B:" << endl;
	//for (int i = 0; i < N; ++i) {
	//	for (int j = 0; j < N; ++j) {
	//		cout << *(b + i * N + j) << "  ";
	//	}
	//	cout << endl;
	//}
	//拷贝显存
	hipMemcpy(p_a, a, sizeof(int) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(p_b, b, sizeof(int) * N * N, hipMemcpyHostToDevice);
	//分配gpu线程块及线程网络组织形式
	dim3 dimGrid(ceil(N / 4.0), ceil(N / 4.0), 1);
	dim3 dimBlock(4, 4, 1);

	//利用event事件在流的执行中添加标记点，打印gpu执行时间
	//cudaEvent_t start1, stop1;
	//cudaEventCreate(&start1);
	//cudaEventCreate(&stop1);
	//cudaEventRecord(start1, 0);
	//MatMul_gpu <<<dimGrid, dimBlock>>> (N, p_a, p_b, p_c);//核函数调用
	//cudaEventRecord(stop1, 0);
	//cudaEventSynchronize(stop1);
	//float time1;
	//cudaEventElapsedTime(&time1, start1, stop1);
	//cout << "gpu time = " << time1 << endl;
	//拷贝计算结果
	//cudaMemcpy(c, p_c, sizeof(int) * N * N, cudaMemcpyDeviceToHost);

	//打印cpu执行时间
	hipEvent_t start2, stop2;
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
	hipEventRecord(start2, 0);
	MatMul_cpu(N, a, b, c);//函数调用
	hipEventRecord(stop2, 0);
	hipEventSynchronize(stop2);
	float time2;
	hipEventElapsedTime(&time2, start2, stop2);
	cout << "cpu time = " << time2 << endl;

	//打印计算结果
	//cout << "A*B:" << endl;
	//for (int i = 0; i < N; ++i) {
	//	for (int j = 0; j < N; ++j) {
	//		cout << *(c + i * N + j) << "  ";
	//	}
	//	cout << endl;
	//}
	//清理内存
	free(a);
	free(b);
	free(c);
	hipFree(p_a);
	hipFree(p_b);
	hipFree(p_c);
	return 0;
}